// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args
// Synthetic test to warn only on device functions umin and umax as unsupported, but not on user defined ones.
// ToDo: change lit testing in order to parse the output.

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

// CHECK: #include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
#include <algorithm>

#define CUDA_LAUNCH(cuda_call,dimGrid,dimBlock, ...) \
    cuda_call<<<dimGrid,dimBlock>>>(__VA_ARGS__);

__global__ void Inc1(float *Ad, float *Bd) {
  int tx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tx < 1) {
    for (int i = 0; i < ITER; ++i) {
      Ad[tx] = Ad[tx] + 1.0f;
      for (int j = 0; j < 256; ++j) {
        Bd[tx] = Ad[tx];
      }
    }
  }
}

int main() {
  float *A, *Ad, *Bd;
  A = new float[LEN];
  for (int i = 0; i < LEN; ++i) {
    A[i] = 0.0f;
  }
  // CHECK: hipError_t status;
  hipError_t status;
  // CHECK: status = hipHostRegister(A, SIZE, hipHostRegisterMapped);
  status = hipHostRegister(A, SIZE, hipHostRegisterMapped);
  // CHECK: hipHostGetDevicePointer(&Ad, A, 0);
  hipHostGetDevicePointer(&Ad, A, 0);
  // CHECK: hipMalloc((void**)&Bd, SIZE);
  hipMalloc((void**)&Bd, SIZE);
  dim3 dimGrid(LEN / 512, 1, 1);
  dim3 dimBlock(512, 1, 1);

  // CHECK: hipLaunchKernelGGL(Inc1, dim3(dimGrid), dim3(dimBlock), 0, 0, Ad, Bd);
  CUDA_LAUNCH(Inc1, dimGrid, dimBlock, Ad, Bd);
}
