
#include <hip/hip_runtime.h>
// RUN: %run_test hipify "%s" "%t" %hipify_args "--skip-excluded-preprocessor-conditional-blocks" %clang_args
// CHECK: #include <hip/hip_runtime.h>

__global__ void axpy_kernel(float a, float* x, float* y) {
  y[threadIdx.x] = a * x[threadIdx.x];
}

void axpy(float a, float* x, float* y) {
float* y_new = nullptr;
#ifdef SOME_MACRO
  y_new = x;
  // CHECK: axpy_kernel <<<1, 1>>> (a, y_new, x);
  axpy_kernel <<<1, 1>>> (a, y_new, x);
#endif

#ifndef SOME_MACRO
  y_new = y;
  // CHECK: hipLaunchKernelGGL(axpy_kernel, dim3(1), dim3(2), 0, 0, a, y_new, x);
  axpy_kernel <<<1, 2>>> (a, y_new, x);
#endif

#ifdef SOME_MACRO
  // CHECK: axpy_kernel <<<1, 3>>> (a, y, x);
  axpy_kernel <<<1, 3>>> (a, y, x);
#else
  // CHECK: hipLaunchKernelGGL(axpy_kernel, dim3(1), dim3(4), 0, 0, a, x, y);
  axpy_kernel <<<1, 4>>> (a, x, y);
#endif

#ifdef SOME_MACRO
  // CHECK: axpy_kernel <<<1, 5>>> (a, y, x);
  axpy_kernel <<<1, 5>>> (a, y, x);
#elif defined SOME_MACRO_1
  // CHECK: axpy_kernel <<<1, 6>>> (a, x, y);
  axpy_kernel <<<1, 6>>> (a, x, y);
#else
  // CHECK: hipLaunchKernelGGL(axpy_kernel, dim3(1), dim3(7), 0, 0, a, x, y);
  axpy_kernel <<<1, 7>>> (a, x, y);
#endif

#ifndef SOME_MACRO
  // CHECK: hipLaunchKernelGGL(axpy_kernel, dim3(1), dim3(8), 0, 0, a, y, x);
  axpy_kernel <<<1, 8>>> (a, y, x);
#elif !defined(SOME_MACRO_1)
  // CHECK: axpy_kernel <<<1, 9>>> (a, x, y);
  axpy_kernel <<<1, 9>>> (a, x, y);
#else
  // CHECK: axpy_kernel <<<1, 10>>> (a, x, y);
  axpy_kernel <<<1, 10>>> (a, x, y);
#endif

}