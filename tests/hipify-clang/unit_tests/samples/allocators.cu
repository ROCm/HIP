// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

#pragma once
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

/**
 * Allocate GPU memory for `count` elements of type `T`.
 */
template<typename T>
static T* gpuMalloc(size_t count) {
    T* ret = nullptr;
    // CHECK: hipMalloc(&ret, count * sizeof(T));
    hipMalloc(&ret, count * sizeof(T));
    return ret;
}

template<typename T>
__global__ void add(int n, T* x, T* y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(int argc, char* argv[]) {
    size_t numElements = 50;
    float *A = gpuMalloc<float>(numElements);
    float* B = gpuMalloc<float>(numElements);
    for (int i = 0; i < numElements; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }
    int blockSize = 512;
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    dim3 dimGrid(numBlocks, 1, 1);
    dim3 dimBlock(blockSize, 1, 1);
    // CHECK: hipLaunchKernelGGL(HIP_KERNEL_NAME(add<float>), dim3(dimGrid), dim3(dimBlock), 0, 0, numElements, A, B);
    add<float><<<dimGrid, dimBlock>>>(numElements, A, B);
    // CHECK: hipDeviceSynchronize();
    hipDeviceSynchronize();
    float maxError = 0.0f;
    for (int i = 0; i < numElements; ++i)
      maxError = fmax(maxError, fabs(B[i] - 3.0f));
    // CHECK: hipFree(A);
    hipFree(A);
    // CHECK: hipFree(B);
    hipFree(B);
    if (maxError == 0.0f)
      return 0;
    return -1;
  }
