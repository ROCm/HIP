// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// To measure effects of memory coalescing. Coalescing.cu
// B. Wilkinson Jan 30, 2011

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BlockSize 16  // Size of blocks, 32 x 32 threads, fixed, used globally

__global__ void gpu_Comput (int *h, int N, int T) {

// Array loaded with global thread ID that acesses that location

	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int threadID = col + row * N;
	int index = row + col * N;  // sequentially down each row

	for (int t = 0; t < T; t++)  // loop to repeat to reduce other time effects
	  h[index] = threadID;  // load array with flattened global thread ID
}

void printArray(int *h, int N) {

	printf("Results of computation, every N/8 numbers, eight numbers\n");

	for (int row = 0; row < N; row += N/8) {
	  for (int col = 0; col < N; col += N/8)
	  printf("%6d  ", h[col + row * N]);
	  printf("\n");
	}
}

int main(int argc, char *argv[])  {

	int T = 100;  // number of iterations, entered at keyboard
	int B = 1;  // number of blocks, entered at keyboard
	char key;

	int *h, *dev_h;  // ptr to array holding numbers on host and device
  // CHECK: hipEvent_t start, stop;
	hipEvent_t start, stop;  // cuda events to measure time
	float elapsed_time_ms1;
  // CHECK: hipEventCreate( &start );
  // CHECK: hipEventCreate( &stop );
	hipEventCreate( &start );
	hipEventCreate( &stop );

/* ------------------------- Keyboard input -----------------------------------*/

do {  // loop to repeat complete program

	printf("Grid Structure 2-D grid, 2-D blocks\n");
	printf("Blocks fixed at 16 x 16 threads, 512 threads, max for compute cap. 1.x\n");
	printf("Enter number of blocks in grid, each dimension, currently %d\n",B);
	scanf("%d",&B);
	printf("Enter number of iterations, currently %d\n",T);
	scanf("%d",&T);

	int N = B * BlockSize;  // size of data array, given input data

	printf("Array size (and total grid-block size) %d x %d\n", N, N);

	dim3 Block(BlockSize, BlockSize);  //Block structure, 32 x 32 max
	dim3 Grid(B, B);  //Grid structure, B x B

/* ------------------------- Allocate Memory-----------------------------------*/

	int size = N * N * sizeof(int);  // number of bytes in total in array
	h = (int*) malloc(size);  // Array on host
  // CHECK: hipMalloc((void**)&dev_h, size);
	hipMalloc((void**)&dev_h, size);  // allocate device memory

/* ------------------------- GPU Computation -----------------------------------*/

  // CHECK: hipEventRecord( start, 0 );
	hipEventRecord( start, 0 );
  // CHECK: hipLaunchKernelGGL(gpu_Comput, dim3(Grid), dim3(Block), 0, 0, dev_h, N, T);
	gpu_Comput<<< Grid, Block >>>(dev_h, N, T);
  // CHECK: hipEventRecord( stop, 0 );
  // CHECK: hipEventSynchronize( stop );
  // CHECK: hipEventElapsedTime( &elapsed_time_ms1, start, stop );
	hipEventRecord( stop, 0 );  // instrument code to measue end time
	hipEventSynchronize( stop );  // wait for all work done by threads
	hipEventElapsedTime( &elapsed_time_ms1, start, stop );
  // CHECK: hipMemcpy(h,dev_h, size ,hipMemcpyDeviceToHost);
	hipMemcpy(h,dev_h, size ,hipMemcpyDeviceToHost);  //Get results to check

	printArray(h,N);
	printf("\nTime to calculate results on GPU: %f ms.\n", elapsed_time_ms1);

/* -------------------------REPEAT PROGRAM INPUT-----------------------------------*/

	printf("\nEnter c to repeat, return to terminate\n");

	scanf("%c",&key);
	scanf("%c",&key);

} while (key == 'c');  // loop of complete program

/* --------------  clean up  ---------------------------------------*/

free(h);
  // CHECK: hipFree(dev_h);
	hipFree(dev_h);
  // CHECK: hipEventDestroy(start);
  // CHECK: hipEventDestroy(stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
