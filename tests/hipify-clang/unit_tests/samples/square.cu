// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

/*
Copyright (c) 2015-2016 Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(cmd) \
{\
    hipError_t error  = cmd;\
    if (error != hipSuccess) { \
        fprintf(stderr, "error: '%s'(%d) at %s:%d\n", hipGetErrorString(error), error,__FILE__, __LINE__); \
        exit(EXIT_FAILURE);\
	  }\
}


/*
 * Square each element in the array A and write to array C.
 */
template <typename T>
__global__ void
vector_square(T *C_d, const T *A_d, size_t N)
{
    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i=offset; i<N; i+=stride) {
        C_d[i] = A_d[i] * A_d[i];
    }
}


int main(int argc, char *argv[])
{
    float *A_d, *C_d;
    float *A_h, *C_h;
    size_t N = 1000000;
    size_t Nbytes = N * sizeof(float);

    // CHECK: hipDeviceProp_t props;
    hipDeviceProp_t props;

    // CHECK: CHECK(hipGetDeviceProperties(&props, 0/*deviceID*/));
    CHECK(hipGetDeviceProperties(&props, 0/*deviceID*/));
    printf ("info: running on device %s\n", props.name);

    printf ("info: allocate host mem (%6.2f MB)\n", 2*Nbytes/1024.0/1024.0);
    A_h = (float*)malloc(Nbytes);

    // CHECK: CHECK(A_h == 0 ? hipErrorMemoryAllocation : hipSuccess );
    CHECK(A_h == 0 ? hipErrorOutOfMemory : hipSuccess );
    C_h = (float*)malloc(Nbytes);

    // CHECK: CHECK(C_h == 0 ? hipErrorMemoryAllocation : hipSuccess );
    CHECK(C_h == 0 ? hipErrorOutOfMemory : hipSuccess );
    // Fill with Phi + i
    for (size_t i=0; i<N; i++)
    {
        A_h[i] = 1.618f + i;
    }

    printf ("info: allocate device mem (%6.2f MB)\n", 2*Nbytes/1024.0/1024.0);
    // CHECK: CHECK(hipMalloc(&A_d, Nbytes));
    // CHECK: CHECK(hipMalloc(&C_d, Nbytes));
    CHECK(hipMalloc(&A_d, Nbytes));
    CHECK(hipMalloc(&C_d, Nbytes));


    printf ("info: copy Host2Device\n");
    // CHECK: CHECK ( hipMemcpy(A_d, A_h, Nbytes, hipMemcpyHostToDevice));
    CHECK ( hipMemcpy(A_d, A_h, Nbytes, hipMemcpyHostToDevice));

    const unsigned blocks = 512;
    const unsigned threadsPerBlock = 256;

    printf ("info: launch 'vector_square' kernel\n");
    // CHECK: hipLaunchKernelGGL(vector_square, dim3(blocks), dim3(threadsPerBlock), 0, 0, C_d, A_d, N);
    vector_square <<<blocks, threadsPerBlock>>> (C_d, A_d, N);

    printf ("info: copy Device2Host\n");
    // CHECK: CHECK ( hipMemcpy(C_h, C_d, Nbytes, hipMemcpyDeviceToHost));
    CHECK ( hipMemcpy(C_h, C_d, Nbytes, hipMemcpyDeviceToHost));

    printf ("info: check result\n");
    for (size_t i=0; i<N; i++)  {
        if (C_h[i] != A_h[i] * A_h[i]) {
            // CHECK: CHECK(hipErrorUnknown);
            CHECK(hipErrorUnknown);
        }
    }
    printf ("PASSED!\n");
}
