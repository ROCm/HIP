// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// Taken from Jonathan Hui blog https://jhui.github.io/2017/03/06/CUDA

#include <stdio.h>
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void staticReverse(int *d, int n)
{
  __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  // Will not conttinue until all threads completed.
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 64;
  int a[n], r[n], d[n];

  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }

  int *d_d;
  // CHECK: hipMalloc(&d_d, n * sizeof(int));
  hipMalloc(&d_d, n * sizeof(int));
  // run version with static shared memory
  // CHECK: hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  // CHECK: hipLaunchKernelGGL(staticReverse, dim3(1), dim3(n), 0, 0, d_d, n);
  staticReverse<<<1,n>>>(d_d, n);
  // CHECK: hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
}
