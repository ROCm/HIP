#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args
// CHECK: #include <hip/hip_runtime.h>
#include <iostream>
// CHECK: #include <hiprand.h>
#include <hiprand.h>
// CHECK: #include <hipcub/hipcub.hpp>
#include <hipcub/hipcub.hpp>

#include <iostream>

template <int BLOCK_WIDTH, int ITEMS_PER_THREAD,
          // CHECK: hipcub::BlockLoadAlgorithm BLOCK_LOAD_ALGO,
          hipcub::BlockLoadAlgorithm BLOCK_LOAD_ALGO,
          // CHECK: hipcub::BlockStoreAlgorithm BLOCK_STORE_ALGO,
          hipcub::BlockStoreAlgorithm BLOCK_STORE_ALGO,
          typename T>
__global__ void sort(const T* data_in, T* data_out){
    // CHECK: typedef hipcub::BlockLoad<T, BLOCK_WIDTH, ITEMS_PER_THREAD, BLOCK_LOAD_ALGO> BlockLoadT;
    typedef hipcub::BlockLoad<T, BLOCK_WIDTH, ITEMS_PER_THREAD, BLOCK_LOAD_ALGO> BlockLoadT;
    // CHECK: typedef hipcub::BlockRadixSort<T, BLOCK_WIDTH, ITEMS_PER_THREAD> BlockRadixSortT;
    typedef hipcub::BlockRadixSort<T, BLOCK_WIDTH, ITEMS_PER_THREAD> BlockRadixSortT;
    // CHECK: typedef hipcub::BlockStore<T, BLOCK_WIDTH, ITEMS_PER_THREAD, BLOCK_STORE_ALGO> BlockStoreT;
    typedef hipcub::BlockStore<T, BLOCK_WIDTH, ITEMS_PER_THREAD, BLOCK_STORE_ALGO> BlockStoreT;
    __shared__ union {
        typename BlockLoadT::TempStorage load;
        typename BlockRadixSortT::TempStorage sort;
        typename BlockStoreT::TempStorage store;
    } tmp_storage;
    T items[ITEMS_PER_THREAD];
    BlockLoadT(tmp_storage.load).Load(data_in + blockIdx.x * BLOCK_WIDTH * ITEMS_PER_THREAD, items);
    __syncthreads();
    BlockRadixSortT(tmp_storage.sort).Sort(items);
    __syncthreads();
    BlockStoreT(tmp_storage.store).Store(data_out + blockIdx.x * BLOCK_WIDTH * ITEMS_PER_THREAD, items);
}

int main() {
    double* d_gpu = NULL;
    double* result_gpu = NULL;
    double* data_sorted = new double[1000*4096];
    // Allocate memory on the GPU
    // CHECK: hipMalloc(&d_gpu, 1000*4096 * sizeof(double));
    hipMalloc(&d_gpu, 1000*4096 * sizeof(double));
    // CHECK: hipMalloc(&result_gpu, 1000*4096 * sizeof(double));
    hipMalloc(&result_gpu, 1000*4096 * sizeof(double));
    // CHECK: hiprandGenerator_t gen;
    hiprandGenerator_t gen;
    // Create generator
    // CHECK: hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // Fill array with random numbers
    // CHECK: hiprandGenerateNormalDouble(gen, d_gpu, 1000*4096, 0.0, 1.0);
    hiprandGenerateNormalDouble(gen, d_gpu, 1000*4096, 0.0, 1.0);
    // Destroy generator
    // CHECK: hiprandDestroyGenerator(gen);
    hiprandDestroyGenerator(gen);
    // Sort data
    // CHECK: hipLaunchKernelGGL(HIP_KERNEL_NAME(sort<512, 8, hipcub::BLOCK_LOAD_TRANSPOSE, hipcub::BLOCK_STORE_TRANSPOSE>), dim3(1000), dim3(512), 0, 0, d_gpu, result_gpu);
    sort<512, 8, hipcub::BLOCK_LOAD_TRANSPOSE, hipcub::BLOCK_STORE_TRANSPOSE><<<1000, 512>>>(d_gpu, result_gpu);
    // CHECK: hipLaunchKernelGGL(HIP_KERNEL_NAME(sort<256, 16, hipcub::BLOCK_LOAD_DIRECT, hipcub::BLOCK_STORE_DIRECT>), dim3(1000), dim3(256), 0, 0, d_gpu, result_gpu);
    sort<256, 16, hipcub::BLOCK_LOAD_DIRECT, hipcub::BLOCK_STORE_DIRECT><<<1000, 256>>>(d_gpu, result_gpu);
    // CHECK: hipMemcpy(data_sorted, result_gpu, 1000*4096*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(data_sorted, result_gpu, 1000*4096*sizeof(double), hipMemcpyDeviceToHost);
    // Write the sorted data to standard out
    for (int i = 0; i < 4095; ++i) {
        std::cout << data_sorted[i] << ", ";
    }
    std::cout << data_sorted[4095] << std::endl;
}
