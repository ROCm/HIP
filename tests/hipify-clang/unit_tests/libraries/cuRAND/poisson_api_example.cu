// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// Taken from: http://docs.nvidia.com/cuda/curand/device-api-overview.html#poisson-api-example
/*
 * This program uses CURAND library for Poisson distribution
 * to simulate queues in store for 16 hours. It shows the
 * difference of using 3 different APIs:
 * - HOST API -arrival of customers is described by Poisson(4)
 * - SIMPLE DEVICE API -arrival of customers is described by
 *     Poisson(4*(sin(x/100)+1)), where x is number of minutes
 *     from store opening time.
 * - ROBUST DEVICE API -arrival of customers is described by:
 *     - Poisson(2) for first 3 hours.
 *     - Poisson(1) for second 3 hours.
 *     - Poisson(3) after 6 hours.  
 */

#include <stdio.h>
#include <stdlib.h>
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include <hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
// CHECK: #include <hiprand.h>
#include <hiprand/hiprand.h>

// CHECK: #define CUDA_CALL(x) do { if((x) != hipSuccess) {
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)
// CHECK: #define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)


#define HOURS 16
#define OPENING_HOUR 7
#define CLOSING_HOUR (OPENING_HOUR + HOURS)

#define access_2D(type, ptr, row, column, pitch)\
    *((type*)((char*)ptr + (row) * pitch) + column)

enum API_TYPE {
    HOST_API = 0,
    SIMPLE_DEVICE_API = 1,
    ROBUST_DEVICE_API = 2,
};

/* global variables */
API_TYPE api;
int report_break;
int cashiers_load_h[HOURS];
__constant__ int cashiers_load[HOURS];
// CHECK: __global__ void setup_kernel(hiprandState_t *state)
__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    // CHECK: hiprand_init(1234, id, 0, &state[id]);
    hiprand_init(1234, id, 0, &state[id]);
}

__inline__ __device__
void update_queue(int id, int min, unsigned int new_customers,
                  unsigned int &queue_length,
                  unsigned int *queue_lengths, size_t pitch)
{
    int balance;
    balance = new_customers - 2 * cashiers_load[(min-1)/60];
    if (balance + (int)queue_length <= 0){
        queue_length = 0;
    }else{
        queue_length += balance;
    }
    /* Store results */
    access_2D(unsigned int, queue_lengths, min-1, id, pitch)
        = queue_length;
}

// CHECK: __global__ void simple_device_API_kernel(hiprandState_t *state,
__global__ void simple_device_API_kernel(hiprandState *state, 
                    unsigned int *queue_lengths, size_t pitch)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int new_customers;
    unsigned int queue_length = 0;
    /* Copy state to local memory for efficiency */
    // CHECK: hiprandState_t localState = state[id];
    hiprandState localState = state[id];
    /* Simulate queue in time */
    for(int min = 1; min <= 60 * HOURS; min++) {
        /* Draw number of new customers depending on API */
        // CHECK: new_customers = hiprand_poisson(&localState,
        new_customers = hiprand_poisson(&localState,
                                4*(sin((float)min/100.0)+1));
        /* Update queue */
        update_queue(id, min, new_customers, queue_length,
                     queue_lengths, pitch);       
    }
    /* Copy state back to global memory */
    state[id] = localState;
}


__global__ void host_API_kernel(unsigned int *poisson_numbers,
                    unsigned int *queue_lengths, size_t pitch)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int new_customers;
    unsigned int queue_length = 0;
    /* Simulate queue in time */
    for(int min = 1; min <= 60 * HOURS; min++) {
        /* Get random number from global memory */
        new_customers = poisson_numbers
                    [blockDim.x * gridDim.x * (min -1) + id];
        /* Update queue */
        update_queue(id, min, new_customers, queue_length,
                     queue_lengths, pitch);
    }
}
// CHECK: __global__ void robust_device_API_kernel(hiprandState_t *state,
// CHECK: hiprandDiscreteDistribution_t poisson_1,
// CHECK: hiprandDiscreteDistribution_t poisson_2,
// CHECK: hiprandDiscreteDistribution_t poisson_3,
__global__ void robust_device_API_kernel(hiprandState *state,
                   hiprandDiscreteDistribution_t poisson_1,
                   hiprandDiscreteDistribution_t poisson_2,
                   hiprandDiscreteDistribution_t poisson_3,
                   unsigned int *queue_lengths, size_t pitch)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int new_customers;
    unsigned int queue_length = 0;
    /* Copy state to local memory for efficiency */
    // CHECK: hiprandState_t localState = state[id];
    hiprandState localState = state[id];
    /* Simulate queue in time */
    /* first 3 hours */
    for(int min = 1; min <= 60 * 3; min++) {
        /* draw number of new customers depending on API */
        new_customers =
                    // CHECK: hiprand_discrete(&localState, poisson_2);
                    hiprand_discrete(&localState, poisson_2);
        /* Update queue */
        update_queue(id, min, new_customers, queue_length,
                                    queue_lengths, pitch);
    }
    /* second 3 hours */
    for(int min = 60 * 3 + 1; min <= 60 * 6; min++) {
        /* draw number of new customers depending on API */
        new_customers =
                    // CHECK: hiprand_discrete(&localState, poisson_1);
                    hiprand_discrete(&localState, poisson_1);
        /* Update queue */
        update_queue(id, min, new_customers, queue_length,
                                    queue_lengths, pitch);       
    }
    /* after 6 hours */
    for(int min = 60 * 6 + 1; min <= 60 * HOURS; min++) {
        /* draw number of new customers depending on API */
        new_customers =
                    // CHECK: hiprand_discrete(&localState, poisson_3);
                    hiprand_discrete(&localState, poisson_3);
        /* Update queue */
        update_queue(id, min, new_customers, queue_length,
                                    queue_lengths, pitch);       
    }
    /* Copy state back to global memory */
    state[id] = localState;
}

/* Set time intervals between reports */
void report_settings()
{
    do{
        printf("Set time intervals between queue reports");
        printf("(in minutes > 0)\n");
        if (scanf("%d", &report_break) == 0) continue;
    }while(report_break <= 0);
}


/* Set number of cashiers each hour */
void add_cachiers(int *cashiers_load)
{
    int i, min, max, begin, end;
    printf("Cashier serves 2 customers per minute...\n");
    for (i = 0; i < HOURS; i++){
        cashiers_load_h[i] = 0;
    }
    while (true){
        printf("Adding cashier...\n");
        min = OPENING_HOUR;
        max = CLOSING_HOUR-1;
        do{
            printf("Set hour that cahier comes (%d-%d)",
                                                min, max);
            printf(" [type 0 to finish adding cashiers]\n");
            if (scanf("%d", &begin) == 0) continue;
        }while (begin > max || (begin < min && begin != 0));
        if (begin == 0) break;
        min = begin+1;
        max = CLOSING_HOUR;
        do{
            printf("Set hour that cahier leaves (%d-%d)",
                                                min, max);
            printf(" [type 0 to finish adding cashiers]\n");
            if (scanf("%d", &end) == 0) continue;
        }while (end > max || (end < min && end != 0));
        if (end == 0) break;
        for (i = begin - OPENING_HOUR;
             i < end   - OPENING_HOUR; i++){
            cashiers_load_h[i]++;
        }
    }
    for (i = OPENING_HOUR; i < CLOSING_HOUR; i++){
        printf("\n%2d:00 - %2d:00     %d cashier",
                i, i+1, cashiers_load_h[i-OPENING_HOUR]);
        if (cashiers_load[i-OPENING_HOUR] != 1) printf("s");
    }
    printf("\n");
}

/* Set API type */
API_TYPE set_API_type()
{
    printf("Choose API type:\n");
    int choose;
    do{
        printf("type 1 for HOST API\n");
        printf("type 2 for SIMPLE DEVICE API\n");
        printf("type 3 for ROBUST DEVICE API\n");
        if (scanf("%d", &choose) == 0) continue;
    }while( choose < 1 || choose > 3);
    switch(choose){
        case 1: return HOST_API;
        case 2: return SIMPLE_DEVICE_API;
        case 3: return ROBUST_DEVICE_API;
        default:
            fprintf(stderr, "wrong API\n");
            return HOST_API;
    }
}

void settings()
{
    add_cachiers(cashiers_load);
    // CHECK: hipMemcpyToSymbol("cashiers_load", cashiers_load_h,
    // CHECK: HOURS * sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("cashiers_load"), cashiers_load_h,
            HOURS * sizeof(int), 0, hipMemcpyHostToDevice);
    report_settings();
    api = set_API_type();
}

void print_statistics(unsigned int *hostResults, size_t pitch)
{
    int min, i, hour, minute;
    unsigned int sum;
    for(min = report_break; min <= 60 * HOURS;
                            min += report_break) {
        sum = 0;
        for(i = 0; i < 64 * 64; i++) {
            sum += access_2D(unsigned int, hostResults,
                                        min-1, i, pitch);
        }
        hour = OPENING_HOUR + min/60;
        minute = min%60;
        printf("%2d:%02d   # of waiting customers = %10.4g |",
                    hour, minute, (float)sum/(64.0 * 64.0));
        printf("  # of cashiers = %d  |  ",
                    cashiers_load_h[(min-1)/60]);
        printf("# of new customers/min ~= ");
        switch (api){
            case HOST_API:
                printf("%2.2f\n", 4.0);
                break;
            case SIMPLE_DEVICE_API:
                printf("%2.2f\n",
                            4*(sin((float)min/100.0)+1));
                break;
            case ROBUST_DEVICE_API:
                if (min <= 3 * 60){
                    printf("%2.2f\n", 2.0);
                }else{
                    if (min <= 6 * 60){
                        printf("%2.2f\n", 1.0);
                    }else{
                        printf("%2.2f\n", 3.0);
                    }
                }
                break;
            default:
                fprintf(stderr, "Wrong API\n");
        }
    }
}


int main(int argc, char *argv[])
{
    int n;
    size_t pitch;
    // CHECK: hiprandState_t *devStates;
    hiprandState *devStates;
    unsigned int *devResults, *hostResults;
    unsigned int *poisson_numbers_d;
    // CHECK: hiprandDiscreteDistribution_t poisson_1, poisson_2;
    // CHECK: hiprandDiscreteDistribution_t poisson_3;
    // CHECK: hiprandGenerator_t gen;
    hiprandDiscreteDistribution_t poisson_1, poisson_2;
    hiprandDiscreteDistribution_t poisson_3;
    hiprandGenerator_t gen;

    /* Setting cashiers, report and API */
    settings();

    /* Allocate space for results on device */
    // CHECK: CUDA_CALL(hipMallocPitch((void **)&devResults, &pitch,
    CUDA_CALL(hipMallocPitch((void **)&devResults, &pitch,
                64 * 64 * sizeof(unsigned int), 60 * HOURS));

    /* Allocate space for results on host */
    hostResults = (unsigned int *)calloc(pitch * 60 * HOURS,
                sizeof(unsigned int));

    /* Allocate space for prng states on device */
    // CHECK: CUDA_CALL(hipMalloc((void **)&devStates, 64 * 64 *
    // CHECK: sizeof(hiprandState_t)));
    CUDA_CALL(hipMalloc((void **)&devStates, 64 * 64 * 
              sizeof(hiprandState)));

    /* Setup prng states */
    if (api != HOST_API){
        // CHECK: hipLaunchKernelGGL(setup_kernel, dim3(64), dim3(64), 0, 0, devStates);
        setup_kernel<<<64, 64>>>(devStates);
    }
    /* Simulate queue  */
    switch (api){
        case HOST_API:
            /* Create pseudo-random number generator */
            // CHECK: CURAND_CALL(hiprandCreateGenerator(&gen,
                                // CHECK: HIPRAND_RNG_PSEUDO_DEFAULT));
            CURAND_CALL(hiprandCreateGenerator(&gen,
                                HIPRAND_RNG_PSEUDO_DEFAULT));
            /* Set seed */
            // CHECK: CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(
                                            gen, 1234ULL));
            /* compute n */
            n = 64 * 64 * HOURS * 60;
            /* Allocate n unsigned ints on device */
            // CHECK: CUDA_CALL(hipMalloc((void **)&poisson_numbers_d,
            CUDA_CALL(hipMalloc((void **)&poisson_numbers_d,
                                n * sizeof(unsigned int)));
            /* Generate n unsigned ints on device */
            // CHECK: CURAND_CALL(hiprandGeneratePoisson(gen,
            CURAND_CALL(hiprandGeneratePoisson(gen,
                                poisson_numbers_d, n, 4.0));
            // CHECK: hipLaunchKernelGGL(host_API_kernel, dim3(64), dim3(64), 0, 0, poisson_numbers_d,
            host_API_kernel<<<64, 64>>>(poisson_numbers_d,
                                        devResults, pitch);
            /* Cleanup */
            // CHECK: CURAND_CALL(hiprandDestroyGenerator(gen));
            CURAND_CALL(hiprandDestroyGenerator(gen));
            break;
        case SIMPLE_DEVICE_API:
            // CHECK: hipLaunchKernelGGL(simple_device_API_kernel, dim3(64), dim3(64), 0, 0, devStates,
            simple_device_API_kernel<<<64, 64>>>(devStates,
                                        devResults, pitch);
            break;
        case ROBUST_DEVICE_API:
            /* Create histograms for Poisson(1) */
            // CHECK: CURAND_CALL(hiprandCreatePoissonDistribution(1.0,
            CURAND_CALL(hiprandCreatePoissonDistribution(1.0,
                                                &poisson_1));
            /* Create histograms for Poisson(2) */
            // CHECK: CURAND_CALL(hiprandCreatePoissonDistribution(2.0,
            CURAND_CALL(hiprandCreatePoissonDistribution(2.0,
                                                &poisson_2));
            /* Create histograms for Poisson(3) */
            // CHECK: CURAND_CALL(hiprandCreatePoissonDistribution(3.0,
            CURAND_CALL(hiprandCreatePoissonDistribution(3.0,
                                                &poisson_3));
            // CHECK: hipLaunchKernelGGL(robust_device_API_kernel, dim3(64), dim3(64), 0, 0, devStates,
            robust_device_API_kernel<<<64, 64>>>(devStates,
                            poisson_1, poisson_2, poisson_3,
                            devResults, pitch);
            /* Cleanup */
            // CHECK: CURAND_CALL(hiprandDestroyDistribution(poisson_1));
            // CHECK: CURAND_CALL(hiprandDestroyDistribution(poisson_2));
            // CHECK: CURAND_CALL(hiprandDestroyDistribution(poisson_3));
            CURAND_CALL(hiprandDestroyDistribution(poisson_1));
            CURAND_CALL(hiprandDestroyDistribution(poisson_2));
            CURAND_CALL(hiprandDestroyDistribution(poisson_3));
            break;
        default:
            fprintf(stderr, "Wrong API\n");
    }
    /* Copy device memory to host */
    // CHECK: CUDA_CALL(hipMemcpy2D(hostResults, pitch, devResults,
    // CHECK: 60 * HOURS, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy2D(hostResults, pitch, devResults,
                pitch, 64 * 64 * sizeof(unsigned int),
                60 * HOURS, hipMemcpyDeviceToHost));
    /* Show result */
    print_statistics(hostResults, pitch);
    /* Cleanup */
    // CHECK: CUDA_CALL(hipFree(devStates));
    // CHECK: CUDA_CALL(hipFree(devResults));
    CUDA_CALL(hipFree(devStates));
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    return EXIT_SUCCESS;
}
