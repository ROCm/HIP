// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include <hipfft.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>

#define DATASIZE 8
#define BATCH 2

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// CHECK: inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  // CHECK: if (code != hipSuccess)
  if (code != hipSuccess)
  {
    // CHECK: fprintf(stderr, "GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
    fprintf(stderr, "GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

int main()
{
  // --- Host side input data allocation and initialization
  // CHECK: hipfftReal *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH * sizeof(hipfftReal));
  hipfftReal *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH * sizeof(hipfftReal));
  for (int i = 0; i<BATCH; i++)
    for (int j = 0; j<DATASIZE; j++) hostInputData[i*DATASIZE + j] = (hipfftReal)(i + 1);

  // --- Device side input data allocation and initialization
  hipfftReal *deviceInputData; gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal)));
  // CHECK: hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);
  hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);

  // --- Host side output data allocation
  hipfftComplex *hostOutputData = (hipfftComplex*)malloc((DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

  // --- Device side output data allocation
  hipfftComplex *deviceOutputData; gpuErrchk(hipMalloc((void**)&deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex)));

  // --- Batched 1D FFTs
  // CHECK: hipfftHandle handle;
  hipfftHandle handle;
  int rank = 1;                           // --- 1D FFTs
  int n[] = { DATASIZE };                 // --- Size of the Fourier transform
  int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
  int idist = DATASIZE, odist = (DATASIZE / 2 + 1); // --- Distance between batches
  int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
  int batch = BATCH;                      // --- Number of batched executions
  // CHECK: hipfftPlanMany(&handle, rank, n,
  hipfftPlanMany(&handle, rank, n,
    inembed, istride, idist,
    // CHECK: onembed, ostride, odist, HIPFFT_R2C, batch);
    onembed, ostride, odist, HIPFFT_R2C, batch);

  // CHECK: hipfftExecR2C(handle, deviceInputData, deviceOutputData);
  hipfftExecR2C(handle, deviceInputData, deviceOutputData);

  // --- Device->Host copy of the results
  // CHECK: gpuErrchk(hipMemcpy(hostOutputData, deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(hostOutputData, deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

  for (int i = 0; i<BATCH; i++)
    for (int j = 0; j<(DATASIZE / 2 + 1); j++)
      printf("%i %i %f %fn", i, j, hostOutputData[i*(DATASIZE / 2 + 1) + j].x, hostOutputData[i*(DATASIZE / 2 + 1) + j].y);

  // CHECK: hipfftDestroy(handle);
  hipfftDestroy(handle);
  // CHECK: gpuErrchk(hipFree(deviceOutputData));
  // CHECK: gpuErrchk(hipFree(deviceInputData));
  gpuErrchk(hipFree(deviceOutputData));
  gpuErrchk(hipFree(deviceInputData));
}
