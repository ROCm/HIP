// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hipblas.h"
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
// CHECK: static __inline__ void modify(hipblasHandle_t handle, float *m, int ldm, int
static __inline__ void modify(hipblasHandle_t handle, float *m, int ldm, int
  n, int p, int q, float alpha, float beta) {
  // CHECK: hipblasSscal(handle, n - p + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
  // CHECK: hipblasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
  hipblasSscal(handle, n - p + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
  hipblasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
}
int main(void) {
  // CHECK: hipError_t cudaStat;
  // CHECK: hipblasStatus_t stat;
  // CHECK: hipblasHandle_t handle;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int i, j;
  float* devPtrA;
  float* a = 0;
  a = (float *)malloc(M * N * sizeof(*a));
  if (!a) {
    printf("host memory allocation failed");
    return EXIT_FAILURE;
  }
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      a[IDX2F(i, j, M)] = (float)((i - 1) * M + j);
    }
  }
  // CHECK: cudaStat = hipMalloc((void**)&devPtrA, M*N * sizeof(*a));
  cudaStat = hipMalloc((void**)&devPtrA, M*N * sizeof(*a));
  // CHECK: if (cudaStat != hipSuccess) {
  if (cudaStat != hipSuccess) {
    printf("device memory allocation failed");
    return EXIT_FAILURE;
  }
  // CHECK: stat = hipblasCreate(&handle);
  stat = hipblasCreate(&handle);
  // CHECK: if (stat != HIPBLAS_STATUS_SUCCESS) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }
  // CHECK: stat = hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
  stat = hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
  // CHECK: if (stat != HIPBLAS_STATUS_SUCCESS) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data download failed");
    // CHECK: hipFree(devPtrA);
    // CHECK: hipblasDestroy(handle);
    hipFree(devPtrA);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  modify(handle, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
  // CHECK: stat = hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
  stat = hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
  // CHECK: if (stat != HIPBLAS_STATUS_SUCCESS) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data upload failed");
    // CHECK: hipFree(devPtrA);
    // CHECK: hipblasDestroy(handle);
    hipFree(devPtrA);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  // CHECK: hipFree(devPtrA);
  // CHECK: hipblasDestroy(handle);
  hipFree(devPtrA);
  hipblasDestroy(handle);
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      printf("%7.0f", a[IDX2F(i, j, M)]);
    }
    printf("\n");
  }
  free(a);
  return EXIT_SUCCESS;
}
