// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

#include <stdio.h>
#include <stdlib.h>
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hipblas.h"
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 6
#define n 4
#define k 5
int main(void) {
  // CHECK: hipError_t cudaStat;
  // CHECK: hipblasStatus_t stat;
  // CHECK: hipblasHandle_t handle;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int i, j;
  float * a;
  float * b;
  float * c;
  a = (float *)malloc(m*k * sizeof(float));
  b = (float *)malloc(k*n * sizeof(float));
  c = (float *)malloc(m*n * sizeof(float));
  int ind = 11;
  for (j = 0; j<k; j++) {
    for (i = 0; i<m; i++) {
      a[IDX2C(i, j, m)] = (float)ind++;
    }
  }
  printf("a:\n");
  for (i = 0; i<m; i++) {
    for (j = 0; j<k; j++) {
      printf(" %5.0f", a[IDX2C(i, j, m)]);
    }
    printf("\n");
  }
  ind = 11;
  for (j = 0; j<n; j++) {
    for (i = 0; i<k; i++) {
      b[IDX2C(i, j, k)] = (float)ind++;
    }
  }
  printf("b:\n");
  for (i = 0; i<k; i++) {
    for (j = 0; j<n; j++) {
      printf(" %5.0f", b[IDX2C(i, j, k)]);
    }
    printf("\n");
  }
  ind = 11;
  for (j = 0; j<n; j++) {
    for (i = 0; i<m; i++) {
      c[IDX2C(i, j, m)] = (float)ind++;
    }
  }
  printf("c:\n");
  for (i = 0; i<m; i++) {
    for (j = 0; j<n; j++) {
      printf(" %5.0f", c[IDX2C(i, j, m)]);
    }
    printf("\n");
  }
  float * d_a;
  float * d_b;
  float * d_c;
  // CHECK: cudaStat = hipMalloc((void **)& d_a, m*k * sizeof(*a));
  // CHECK: cudaStat = hipMalloc((void **)& d_b, k*n * sizeof(*b));
  // CHECK: cudaStat = hipMalloc((void **)& d_c, m*n * sizeof(*c));
  cudaStat = hipMalloc((void **)& d_a, m*k * sizeof(*a));
  cudaStat = hipMalloc((void **)& d_b, k*n * sizeof(*b));
  cudaStat = hipMalloc((void **)& d_c, m*n * sizeof(*c));
  // CHECK: stat = hipblasCreate(&handle);
  stat = hipblasCreate(&handle);
  // CHECK: stat = hipblasSetMatrix(m, k, sizeof(*a), a, m, d_a, m);
  // CHECK: stat = hipblasSetMatrix(k, n, sizeof(*b), b, k, d_b, k);
  // CHECK: stat = hipblasSetMatrix(m, n, sizeof(*c), c, m, d_c, m);
  stat = hipblasSetMatrix(m, k, sizeof(*a), a, m, d_a, m);
  stat = hipblasSetMatrix(k, n, sizeof(*b), b, k, d_b, k);
  stat = hipblasSetMatrix(m, n, sizeof(*c), c, m, d_c, m);
  float al = 1.0f;
  float bet = 1.0f;
  // CHECK: stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, d_a, m, d_b, k, &bet, d_c, m);
  stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, d_a, m, d_b, k, &bet, d_c, m);
  // CHECK: stat = hipblasGetMatrix(m, n, sizeof(*c), d_c, m, c, m);
  stat = hipblasGetMatrix(m, n, sizeof(*c), d_c, m, c, m);
  printf("c after Sgemm :\n");
  for (i = 0; i<m; i++) {
    for (j = 0; j<n; j++) {
      printf(" %7.0f", c[IDX2C(i, j, m)]);
    }
    printf("\n");
  }
  // CHECK: hipFree(d_a);
  // CHECK: hipFree(d_b);
  // CHECK: hipFree(d_c);
  // CHECK: hipblasDestroy(handle);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipblasDestroy(handle);
  free(a);
  free(b);
  free(c);
  return EXIT_SUCCESS;
}
