// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>
// CHECK: #include <iostream>
// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
// CHECK: #include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"
#include "hipblas.h"
#include <stdio.h>
