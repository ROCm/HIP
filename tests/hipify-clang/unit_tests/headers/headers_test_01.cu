// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK-NOT: #include <cuda_runtime.h>
// CHECK: #include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
