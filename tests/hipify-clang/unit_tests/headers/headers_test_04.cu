// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK-NEXT: #include <stdio.h>
// CHECK-NEXT: #include <iostream>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
// CHECK-NOT: #include <hip/hip_runtime.h>
int main(int argc, char* argv[]) {
  return 0;
}

