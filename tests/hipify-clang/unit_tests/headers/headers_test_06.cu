// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hipblas.h>
// CHECK-NOT: #include <hipblas.h>
// CHECK: #include <stdio.h>
#include <hipblas.h>
#include <hipblas.h>
#include <stdio.h>
