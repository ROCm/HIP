// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
// CHECK: #include <stdio.h>
#include "hipblas.h"
#include "hipblas.h"
#include <stdio.h>
