// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK: #include <memory>

// CHECK-NOT: #include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>

// CHECK: #include "hip/hip_runtime_api.h"
// CHECK: #include "hip/hip/channel_descriptor.h"
// CHECK: #include "hip/hip/device_functions.h"
// CHECK: #include "hip/hip/driver_types.h"
// CHECK: #include "hip/hip_complex.h"
// CHECK: #include "hip/hip_fp16.h"
// CHECK: #include "hip/hip_texture_types.h"
// CHECK: #include "hip/hip_vector_types.h"

// CHECK: #include <iostream>

// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"

// CHECK: #include <stdio.h>

// CHECK: #include "hiprand.h"
// CHECK: #include "hiprand_kernel.h"

// CHECK: #include <algorithm>

// CHECK-NOT: #include "hiprand.h"
// CHECK-NOT: #include "hiprand_kernel.h"
// CHECK-NOT: #include "hiprand_discrete.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_mtgp32_host.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "rocrand/rocrand_mtgp32_11213.h"
// CHECK-NOT: #include "hiprand_normal.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand_poisson.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand_uniform.h"

// CHECK: #include <string>

// CHECK: #include "hipfft.h"
// CHECK: #include "hipsparse.h"

#include <hip/hip_runtime.h>

#include <memory>

#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include "hip/channel_descriptor.h"
#include "hip/device_functions.h"
#include "hip/driver_types.h"
#include "hip/hip_complex.h"
#include "hip/hip_fp16.h"
#include "hip/hip_texture_types.h"
#include "hip/hip_vector_types.h"

#include <iostream>

#include "hipblas.h"
#include "hipblas.h"

#include <stdio.h>

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <algorithm>

#include "hiprand_discrete.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_mtgp32_host.h"
#include "hiprand/hiprand_kernel.h"
#include "rocrand/rocrand_mtgp32_11213.h"
#include "hiprand_normal.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand_poisson.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand_uniform.h"

#include <string>

#include "hipfft/hipfft.h"

#include "hipsparse.h"
