#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %cuda_args
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }
    // CHECK: hipError_t error;
    hipError_t error;
    // CHECK: error = hipMalloc((void **) &d_A, mem_size_A);
    error = hipMalloc((void **) &d_A, mem_size_A);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        // CHECK: printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    // CHECK: error = hipMalloc((void **) &d_B, mem_size_B);
    error = hipMalloc((void **) &d_B, mem_size_B);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        // CHECK: printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    // CHECK: error = hipMalloc((void **) &d_C, mem_size_C);
    error = hipMalloc((void **) &d_C, mem_size_C);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        // CHECK: printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    // CHECK: error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        // CHECK: printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    // CHECK: error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        // CHECK: printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16)
    {
        // CHECK: hipLaunchKernelGGL(matrixMulCUDA<16>, dim3(grid), dim3(threads), 0, 0, d_C, d_A, d_B, dimsA.x, dimsB.x);
        matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else
    {
        // CHECK: hipLaunchKernelGGL(matrixMulCUDA<32>, dim3(grid), dim3(threads), 0, 0, d_C, d_A, d_B, dimsA.x, dimsB.x);
        matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("done\n");
    // CHECK: hipDeviceSynchronize();
    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    // CHECK: hipEvent_t start;
    hipEvent_t start;
    // CHECK: error = hipEventCreate(&start);
    error = hipEventCreate(&start);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    // CHECK: hipEvent_t stop;
    hipEvent_t stop;
    // CHECK: error = hipEventCreate(&stop);
    error = hipEventCreate(&stop);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    // CHECK: error = hipEventRecord(stop, NULL);
    error = hipEventRecord(start, NULL);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++)
    {
        if (block_size == 16)
        {
            matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);
    // CHECK: if (error != hipSuccess)
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6 ; // machine zero

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    // CHECK: hipFree(d_A);
    // CHECK: hipFree(d_B);
    // CHECK: hipFree(d_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    int block_size = 32;

    dim3 dimsA(5*2*block_size, 5*2*block_size, 1);
    dim3 dimsB(5*4*block_size, 5*2*block_size, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA"))
    {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA"))
    {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB"))
    {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB"))
    {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}
