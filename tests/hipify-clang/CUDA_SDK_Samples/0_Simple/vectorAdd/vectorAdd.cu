#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %cuda_args
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    // CHECK: hipError_t err = hipSuccess;
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    // CHECK: err = hipMalloc((void **)&d_A, size);
    err = hipMalloc((void **)&d_A, size);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    // CHECK: err = hipMalloc((void **)&d_B, size);
    err = hipMalloc((void **)&d_B, size);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    // CHECK: err = hipMalloc((void **)&d_C, size);
    err = hipMalloc((void **)&d_C, size);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    // CHECK: err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // CHECK: err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    // CHECK: hipLaunchKernelGGL(vectorAdd, dim3(blocksPerGrid), dim3(threadsPerBlock), 0, 0, d_A, d_B, d_C, numElements);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    // CHECK: err = hipGetLastError();
    err = hipGetLastError();
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    // CHECK: err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        // CHECK: fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    // CHECK: err = hipFree(d_A);
    err = hipFree(d_A);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // CHECK: err = hipFree(d_B);
    err = hipFree(d_B);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // CHECK: err = hipFree(d_C);
    err = hipFree(d_C);
    // CHECK: if (err != hipSuccess)
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

