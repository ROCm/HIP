#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %cuda_args

/*
* This software contains source code provided by NVIDIA Corporation.
*/

/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <iostream>
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

const int manualBlockSize = 32;

////////////////////////////////////////////////////////////////////////////////
// Test kernel
//
// This kernel squares each array element. Each thread addresses
// himself with threadIdx and blockIdx, so that it can handle any
// execution configuration, including anything the launch configurator
// API suggests.
////////////////////////////////////////////////////////////////////////////////
__global__ void square(int *array, int arrayCount)
{
    // CHECK: HIP_DYNAMIC_SHARED(int, dynamicSmem);
    extern __shared__ int dynamicSmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < arrayCount) {
        array[idx] *= array[idx];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Potential occupancy calculator
//
// The potential occupancy is calculated according to the kernel and
// execution configuration the user desires. Occupancy is defined in
// terms of active blocks per multiprocessor, and the user can convert
// it to other metrics.
//
// This wrapper routine computes the occupancy of kernel, and reports
// it in terms of active warps / maximum warps per SM.
////////////////////////////////////////////////////////////////////////////////
static double reportPotentialOccupancy(void *kernel, int blockSize, size_t dynamicSMem)
{
    int device;
    // CHECK: hipDeviceProp_t prop;
    hipDeviceProp_t prop;

    int numBlocks;
    int activeWarps;
    int maxWarps;

    double occupancy;

    // CHECK: checkCudaErrors(hipGetDevice(&device));
    // CHECK: checkCudaErrors(hipGetDeviceProperties(&prop, device));
    // CHECK: checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipGetDeviceProperties(&prop, device));

    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
                        &numBlocks,
                        kernel,
                        blockSize,
                        dynamicSMem));

    // CHECK: activeWarps = numBlocks * blockSize / prop.hipWarpSize;
    // CHECK: maxWarps = prop.maxThreadsPerMultiProcessor / prop.hipWarpSize;
    activeWarps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    occupancy = (double)activeWarps / maxWarps;

    return occupancy;
}

////////////////////////////////////////////////////////////////////////////////
// Occupancy-based launch configurator
//
// The launch configurator, hipOccupancyMaxPotentialBlockSize and
// hipOccupancyMaxPotentialBlockSizeVariableSMem, suggests a block
// size that achieves the best theoretical occupancy. It also returns
// the minimum number of blocks needed to achieve the occupancy on the
// whole device.
//
// This launch configurator is purely occupancy-based. It doesn't
// translate directly to performance, but the suggestion should
// nevertheless be a good starting point for further optimizations.
//
// This function configures the launch based on the "automatic"
// argument, records the runtime, and reports occupancy and runtime.
////////////////////////////////////////////////////////////////////////////////
static int launchConfig(int *array, int arrayCount, bool automatic)
{
    int blockSize;
    int minGridSize;
    int gridSize;
    size_t dynamicSMemUsage = 0;
    // CHECK: hipEvent_t start;
    // CHECK: hipEvent_t end;
    hipEvent_t start;
    hipEvent_t end;

    float elapsedTime;
    
    double potentialOccupancy;
    // CHECK: checkCudaErrors(hipEventCreate(&start));
    // CHECK: checkCudaErrors(hipEventCreate(&end));
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    if (automatic) {
        // CHECK: checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                            &minGridSize,
                            &blockSize,
                            (void*)square,
                            dynamicSMemUsage,
                            arrayCount));

        std::cout << "Suggested block size: " << blockSize << std::endl
                  << "Minimum grid size for maximum occupancy: " << minGridSize << std::endl;
    } else {
        // This block size is too small. Given limited number of
        // active blocks per multiprocessor, the number of active
        // threads will be limited, and thus unable to achieve maximum
        // occupancy.
        //
        blockSize = manualBlockSize;
    }

    // Round up
    //
    gridSize = (arrayCount + blockSize - 1) / blockSize;

    // Launch and profile
    //
    // CHECK: checkCudaErrors(hipEventRecord(start));
    // CHECK: hipLaunchKernelGGL(square, dim3(gridSize), dim3(blockSize), dynamicSMemUsage, 0, array, arrayCount);
    // CHECK: checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipEventRecord(start));
    square<<<gridSize, blockSize, dynamicSMemUsage>>>(array, arrayCount);
    checkCudaErrors(hipEventRecord(end));
    // CHECK: checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipDeviceSynchronize());

    // Calculate occupancy
    //
    potentialOccupancy = reportPotentialOccupancy((void*)square, blockSize, dynamicSMemUsage);

    std::cout << "Potential occupancy: " << potentialOccupancy * 100 << "%" << std::endl;

    // Report elapsed time
    //
    // CHECK: checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
    std::cout << "Elapsed time: " << elapsedTime << "ms" << std::endl;
    
    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// The test
//
// The test generates an array and squares it with a CUDA kernel, then
// verifies the result.
////////////////////////////////////////////////////////////////////////////////
static int test(bool automaticLaunchConfig, const int count = 1000000)
{
    int *array;
    int *dArray;
    int size = count * sizeof(int);

    array = new int[count];

    for (int i = 0; i < count; i += 1) {
        array[i] = i;
    }
    // CHECK: checkCudaErrors(hipMalloc(&dArray, size));
    // CHECK: checkCudaErrors(hipMemcpy(dArray, array, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&dArray, size));
    checkCudaErrors(hipMemcpy(dArray, array, size, hipMemcpyHostToDevice));

    for (int i = 0; i < count; i += 1) {
        array[i] = 0;
    }

    launchConfig(dArray, count, automaticLaunchConfig);
    // CHECK: checkCudaErrors(hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost));
    // CHECK: checkCudaErrors(hipFree(dArray));
    checkCudaErrors(hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(dArray));

    // Verify the return data
    //
    for (int i = 0; i < count; i += 1) {
        if (array[i] != i * i) {
            std::cout << "element " << i << " expected " << i * i << " actual " << array[i] << std::endl;
            return 1;
        }
    }
    delete[] array;

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Sample Main
//
// The sample runs the test with manually configured launch and
// automatically configured launch, and reports the occupancy and
// performance.
////////////////////////////////////////////////////////////////////////////////
int main()
{
    int status;

    std::cout << "starting Simple Occupancy" << std::endl << std::endl;

    std::cout << "[ Manual configuration with " << manualBlockSize
              << " threads per block ]" << std::endl;

    status = test(false);
    if (status) {
        std::cerr << "Test failed\n" << std::endl;
        return -1;
    }

    std::cout << std::endl;

    std::cout << "[ Automatic, occupancy-based configuration ]" << std::endl;
    status = test(true);
    if (status) {
        std::cerr << "Test failed\n" << std::endl;
        return -1;
    }        

    std::cout << std::endl;
    std::cout << "Test PASSED\n" << std::endl;
    
    return 0;
}
