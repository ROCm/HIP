#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %cuda_args

/*
* This software contains source code provided by NVIDIA Corporation.
*/

/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* pitchLinearTexture
*
* This example demonstrates how to use textures bound to pitch linear memory.
* It performs a shift of matrix elements using wrap addressing mode (aka
* periodic boundary conditions) on two arrays, a pitch linear and a CUDA array,
* in order to highlight the differences in using each.
*
* Textures binding to pitch linear memory is a new feature in CUDA 2.2,
* and allows use of texture features such as wrap addressing mode and
* filtering which are not possible with textures bound to regular linear memory
*/

// includes, system
#include <stdio.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define NUM_REPS 100  // number of repetitions performed
#define TILE_DIM 16   // tile/block size

const char *sSDKsample = "simplePitchLinearTexture";

////////////////////////////////////////////////////////////////////////////////
// Texture references
// CHECK: texture<float, 2, hipReadModeElementType> texRefPL;
// CHECK: texture<float, 2, hipReadModeElementType> texRefArray;
texture<float, 2, hipReadModeElementType> texRefPL;
texture<float, 2, hipReadModeElementType> texRefArray;

// Auto-Verification Code
bool bTestResult = true;

////////////////////////////////////////////////////////////////////////////////
// NB: (1) The second argument "pitch" is in elements, not bytes
//     (2) normalized coordinates are used (required for wrap address mode)
////////////////////////////////////////////////////////////////////////////////
//! Shifts matrix elements using pitch linear array
//! @param odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void shiftPitchLinear(float *odata,
                                 int pitch,
                                 int width,
                                 int height,
                                 int shiftX,
                                 int shiftY)
{
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    odata[yid * pitch + xid] = tex2D(texRefPL,
                                     (xid + shiftX) / (float) width,
                                     (yid + shiftY) / (float) height);
}

////////////////////////////////////////////////////////////////////////////////
//! Shifts matrix elements using regular array
//! @param odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void shiftArray(float *odata,
                           int pitch,
                           int width,
                           int height,
                           int shiftX,
                           int shiftY)
{
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    odata[yid * pitch + xid] = tex2D(texRefArray,
                                     (xid + shiftX) / (float) width,
                                     (yid + shiftY) / (float) height);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n\n", sSDKsample);

    runTest(argc, argv);

    printf("%s completed, returned %s\n",
           sSDKsample,
           bTestResult ? "OK" : "ERROR!");
    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
    // Set array size
    const int nx = 2048;
    const int ny = 2048;

    // Setup shifts applied to x and y data
    const int x_shift = 5;
    const int y_shift = 7;

    if ((nx % TILE_DIM != 0)  || (ny % TILE_DIM != 0))
    {
        printf("nx and ny must be multiples of TILE_DIM\n");
        exit(EXIT_FAILURE);
    }

    // Setup execution configuration parameters
    dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM), dimBlock(TILE_DIM, TILE_DIM);

    // This will pick the best possible CUDA capable device
    int devID = findCudaDevice(argc, (const char **)argv);

    // CUDA events for timing
    // CHECK: hipEvent_t start, stop;
    // CHECK: hipEventCreate(&start);
    // CHECK: hipEventCreate(&stop);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host allocation and initialization
    float *h_idata = (float *) malloc(sizeof(float) * nx * ny);
    float *h_odata = (float *) malloc(sizeof(float) * nx * ny);
    float *gold = (float *) malloc(sizeof(float) * nx * ny);

    for (int i = 0; i < nx * ny; ++i)
    {
        h_idata[i] = (float) i;
    }

    // Device memory allocation
    // Pitch linear input data
    float *d_idataPL;
    size_t d_pitchBytes;
    // CHECK: checkCudaErrors(hipMallocPitch((void **) &d_idataPL,
    checkCudaErrors(hipMallocPitch((void **) &d_idataPL,
                                    &d_pitchBytes,
                                    nx * sizeof(float),
                                    ny));

    // Array input data
    // CHECK: hipArray *d_idataArray;
    // CHECK: hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray *d_idataArray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    // CHECK: checkCudaErrors(hipMallocArray(&d_idataArray, &channelDesc, nx, ny));
    checkCudaErrors(hipMallocArray(&d_idataArray, &channelDesc, nx, ny));

    // Pitch linear output data
    float *d_odata;
    // CHECK: checkCudaErrors(hipMallocPitch((void **) &d_odata,
    checkCudaErrors(hipMallocPitch((void **) &d_odata,
                                    &d_pitchBytes,
                                    nx * sizeof(float),
                                    ny));

    // Copy host data to device
    // Pitch linear
    size_t h_pitchBytes = nx * sizeof(float);
    // CHECK: checkCudaErrors(hipMemcpy2D(d_idataPL,
    checkCudaErrors(hipMemcpy2D(d_idataPL,
                                 d_pitchBytes,
                                 h_idata,
                                 h_pitchBytes,
                                 nx * sizeof(float),
                                 ny,
                                 // CHECK: hipMemcpyHostToDevice));
                                 hipMemcpyHostToDevice));

    // Array
    // CHECK: checkCudaErrors(hipMemcpyToArray(d_idataArray,
    checkCudaErrors(hipMemcpyToArray(d_idataArray,
                                      0,
                                      0,
                                      h_idata,
                                      nx * ny * sizeof(float),
                                      // CHECK: hipMemcpyHostToDevice));
                                      hipMemcpyHostToDevice));

    // Bind texture to memory
    // Pitch linear
    texRefPL.normalized = 1;
    // CHECK: texRefPL.filterMode = hipFilterModePoint;
    // CHECK: texRefPL.addressMode[0] = hipAddressModeWrap;
    // CHECK: texRefPL.addressMode[1] = hipAddressModeWrap;
    texRefPL.filterMode = hipFilterModePoint;
    texRefPL.addressMode[0] = hipAddressModeWrap;
    texRefPL.addressMode[1] = hipAddressModeWrap;
    // CHECK: checkCudaErrors(hipBindTexture2D(0,
    checkCudaErrors(hipBindTexture2D(0,
                                      &texRefPL,
                                      d_idataPL,
                                      &channelDesc,
                                      nx,
                                      ny,
                                      d_pitchBytes));

    // Array
    texRefArray.normalized = 1;
    // CHECK: texRefArray.filterMode = hipFilterModePoint;
    // CHECK: texRefArray.addressMode[0] = hipAddressModeWrap;
    // CHECK: texRefArray.addressMode[1] = hipAddressModeWrap;
    texRefArray.filterMode = hipFilterModePoint;
    texRefArray.addressMode[0] = hipAddressModeWrap;
    texRefArray.addressMode[1] = hipAddressModeWrap;
    // CHECK: checkCudaErrors(hipBindTextureToArray(texRefArray,
    checkCudaErrors(hipBindTextureToArray(texRefArray,
                                           d_idataArray,
                                           channelDesc));

    // Reference calculation
    for (int j = 0; j < ny; ++j)
    {
        int jshift = (j + y_shift) % ny;

        for (int i = 0; i < nx; ++i)
        {
            int ishift = (i + x_shift) % nx;
            gold[j * nx + i] = h_idata[jshift * nx + ishift];
        }
    }

    // Run ShiftPitchLinear kernel
    // CHECK: checkCudaErrors(hipMemset2D(d_odata,
    checkCudaErrors(hipMemset2D(d_odata,
                                 d_pitchBytes,
                                 0,
                                 nx * sizeof(float),
                                 ny));
    // CHECK: checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start, 0));

    for (int i = 0; i < NUM_REPS; ++i)
    {
        // CHECK: hipLaunchKernelGGL(shiftPitchLinear, dim3(dimGrid), dim3(dimBlock), 0, 0, d_odata,
        shiftPitchLinear<<<dimGrid, dimBlock>>>(d_odata,
         (int)(d_pitchBytes / sizeof(float)),
         nx,
         ny,
         x_shift,
         y_shift);
    }
    // CHECK: checkCudaErrors(hipEventRecord(stop, 0));
    // CHECK: checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float timePL;
    // CHECK: checkCudaErrors(hipEventElapsedTime(&timePL, start, stop));
    checkCudaErrors(hipEventElapsedTime(&timePL, start, stop));

    // Check results
    // CHECK: checkCudaErrors(hipMemcpy2D(h_odata,
    checkCudaErrors(hipMemcpy2D(h_odata,
                                 h_pitchBytes,
                                 d_odata,
                                 d_pitchBytes,
                                 nx * sizeof(float),
                                 ny,
                                 // CHECK: hipMemcpyDeviceToHost));
                                 hipMemcpyDeviceToHost));

    bool res = compareData(gold, h_odata, nx*ny, 0.0f, 0.15f);

    bTestResult = true;

    if (res == false)
    {
        printf("*** shiftPitchLinear failed ***\n");
        bTestResult = false;
    }

    // Run ShiftArray kernel
    // CHECK: checkCudaErrors(hipMemset2D(d_odata,
    checkCudaErrors(hipMemset2D(d_odata,
                                 d_pitchBytes,
                                 0,
                                 nx * sizeof(float),
                                 ny));
    // CHECK: checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start, 0));

    for (int i = 0; i < NUM_REPS; ++i)
    {
        // CHECK: hipLaunchKernelGGL(shiftArray, dim3(dimGrid), dim3(dimBlock), 0, 0, d_odata,
        shiftArray<<<dimGrid, dimBlock>>>(d_odata,
         (int)(d_pitchBytes / sizeof(float)),
         nx,
         ny,
         x_shift,
         y_shift);
    }
    // CHECK: checkCudaErrors(hipEventRecord(stop, 0));
    // CHECK: checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float timeArray;
    // CHECK: checkCudaErrors(hipEventElapsedTime(&timeArray, start, stop));
    checkCudaErrors(hipEventElapsedTime(&timeArray, start, stop));

    // Check results
    // CHECK: checkCudaErrors(hipMemcpy2D(h_odata,
    checkCudaErrors(hipMemcpy2D(h_odata,
                                 h_pitchBytes,
                                 d_odata,
                                 d_pitchBytes,
                                 nx * sizeof(float),
                                 ny,
                                 // CHECK: hipMemcpyDeviceToHost));
                                 hipMemcpyDeviceToHost));
    res = compareData(gold, h_odata, nx*ny, 0.0f, 0.15f);

    if (res == false)
    {
        printf("*** shiftArray failed ***\n");
        bTestResult = false;
    }

    float bandwidthPL =
        2.f * 1000.f * nx * ny * sizeof(float) /
        (1.e+9f) / (timePL / NUM_REPS);
    float bandwidthArray =
        2.f * 1000.f * nx * ny * sizeof(float) /
        (1.e+9f) / (timeArray / NUM_REPS);

    printf("\nBandwidth (GB/s) for pitch linear: %.2e; for array: %.2e\n",
           bandwidthPL, bandwidthArray);

    float fetchRatePL =
        nx * ny / 1.e+6f / (timePL / (1000.0f * NUM_REPS));
    float fetchRateArray =
        nx * ny / 1.e+6f / (timeArray / (1000.0f * NUM_REPS));

    printf("\nTexture fetch rate (Mpix/s) for pitch linear: "
           "%.2e; for array: %.2e\n\n",
           fetchRatePL, fetchRateArray);

    // Cleanup
    free(h_idata);
    free(h_odata);
    free(gold);
    // CHECK: checkCudaErrors(hipUnbindTexture(texRefPL));
    // CHECK: checkCudaErrors(hipUnbindTexture(texRefArray));
    // CHECK: checkCudaErrors(hipFree(d_idataPL));
    // CHECK: checkCudaErrors(hipFreeArray(d_idataArray));
    // CHECK: checkCudaErrors(hipFree(d_odata));
    checkCudaErrors(hipUnbindTexture(texRefPL));
    checkCudaErrors(hipUnbindTexture(texRefArray));
    checkCudaErrors(hipFree(d_idataPL));
    checkCudaErrors(hipFreeArray(d_idataArray));
    checkCudaErrors(hipFree(d_odata));
    // CHECK: checkCudaErrors(hipEventDestroy(start));
    // CHECK: checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
}
