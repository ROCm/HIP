#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %cuda_args

/*
 * This software contains source code provided by NVIDIA Corporation.
*/

/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample demonstrates how use texture fetches in CUDA
 *
 * This sample takes an input PGM image (image_filename) and generates
 * an output PGM image (image_filename_out).  This CUDA kernel performs
 * a simple 2D transform (rotation) on the texture coordinates (u,v).
 */

// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

// Define the files that are to be save and the reference images for validation
const char *imageFilename = "lena_bw.pgm";
const char *refFilename   = "ref_rotated.pgm";

const char *sampleName = "simpleTexture";

////////////////////////////////////////////////////////////////////////////////
// Constants
const float angle = 0.5f;        // angle to rotate image by (in radians)

// Texture reference for 2D float texture
// CHECK: texture<float, 2, hipReadModeElementType> tex;
texture<float, 2, hipReadModeElementType> tex;

// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transformKernel(float *outputData,
                                int width,
                                int height,
                                float theta)
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = (float)x - (float)width/2; 
    float v = (float)y - (float)height/2; 
    float tu = u*cosf(theta) - v*sinf(theta); 
    float tv = v*cosf(theta) + u*sinf(theta); 

    tu /= (float)width; 
    tv /= (float)height; 

    // read from texture and write to global memory
    outputData[y*width + x] = tex2D(tex, tu+0.5f, tv+0.5f);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n", sampleName);

    // Process command-line arguments
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **) argv, "input"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "input",
                                     (char **) &imageFilename);

            if (checkCmdLineFlag(argc, (const char **) argv, "reference"))
            {
                getCmdLineArgumentString(argc,
                                         (const char **) argv,
                                         "reference",
                                         (char **) &refFilename);
            }
            else
            {
                printf("-input flag should be used with -reference flag");
                exit(EXIT_FAILURE);
            }
        }
        else if (checkCmdLineFlag(argc, (const char **) argv, "reference"))
        {
            printf("-reference flag should be used with -input flag");
            exit(EXIT_FAILURE);
        }
    }

    runTest(argc, argv);

    printf("%s completed, returned %s\n",
           sampleName,
           testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
    int devID = findCudaDevice(argc, (const char **) argv);

    // load image from disk
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    //Load reference image from image (output)
    float *hDataRef = (float *) malloc(size);
    char *refPath = sdkFindFilePath(refFilename, argv[0]);

    if (refPath == NULL)
    {
        printf("Unable to find reference image file: %s\n", refFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(refPath, &hDataRef, &width, &height);

    // Allocate device memory for result
    float *dData = NULL;
    // CHECK: checkCudaErrors(hipMalloc((void **) &dData, size));
    checkCudaErrors(hipMalloc((void **) &dData, size));

    // Allocate array and copy image data
    // CHECK: hipChannelFormatDesc channelDesc =
    hipChannelFormatDesc channelDesc =
    // CHECK: hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    // CHECK: hipArray *cuArray;
    hipArray *cuArray;
    // CHECK: checkCudaErrors(hipMallocArray(&cuArray,
    checkCudaErrors(hipMallocArray(&cuArray,
                                    &channelDesc,
                                    width,
                                    height));
    // CHECK: checkCudaErrors(hipMemcpyToArray(cuArray,
    checkCudaErrors(hipMemcpyToArray(cuArray,
                                      0,
                                      0,
                                      hData,
                                      size,
                                      hipMemcpyHostToDevice));

    // Set texture parameters
    // CHECK: tex.addressMode[0] = hipAddressModeWrap;
    // CHECK: tex.addressMode[1] = hipAddressModeWrap;
    // CHECK: tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    // CHECK: checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));
    checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // Warmup
    // CHECK: hipLaunchKernelGGL(transformKernel, dim3(dimGrid), dim3(dimBlock), 0, 0, dData, width, height, angle);
    transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height, angle);
    // CHECK: checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipDeviceSynchronize());
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    // CHECK: hipLaunchKernelGGL(transformKernel, dim3(dimGrid), dim3(dimBlock), 0, 0, dData, width, height, angle);
    transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height, angle);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");
    // CHECK: checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    printf("%.2f Mpixels/sec\n",
           (width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

    // Allocate mem for the result on host side
    float *hOutputData = (float *) malloc(size);
    // copy result from device to host
    // CHECK: checkCudaErrors(hipMemcpy(hOutputData,
    checkCudaErrors(hipMemcpy(hOutputData,
                               dData,
                               size,
    // CHECK: hipMemcpyDeviceToHost));
                               hipMemcpyDeviceToHost));

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
    sdkSavePGM(outputFilename, hOutputData, width, height);
    printf("Wrote '%s'\n", outputFilename);

    // Write regression file if necessary
    if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
    {
        // Write file for regression test
        sdkWriteFile<float>("./data/regression.dat",
                            hOutputData,
                            width*height,
                            0.0f,
                            false);
    }
    else
    {
        // We need to reload the data from disk,
        // because it is inverted upon output
        sdkLoadPGM(outputFilename, &hOutputData, &width, &height);

        printf("Comparing files\n");
        printf("\toutput:    <%s>\n", outputFilename);
        printf("\treference: <%s>\n", refPath);

        testResult = compareData(hOutputData,
                                 hDataRef,
                                 width*height,
                                 MAX_EPSILON_ERROR,
                                 0.15f);
    }
    // CHECK: checkCudaErrors(hipFree(dData));
    // CHECK: checkCudaErrors(hipFreeArray(cuArray));
    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFreeArray(cuArray));
    free(imagePath);
    free(refPath);
}
